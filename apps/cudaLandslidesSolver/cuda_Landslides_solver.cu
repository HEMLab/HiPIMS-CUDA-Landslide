#include "hip/hip_runtime.h"
// ======================================================================================
// Name                :    GeoClasses : Generic Geophysical Flow Modelling Framework
// Description         :    This code pack provides a generic framework for developing 
//                          Geophysical CFD software.
// ======================================================================================
// Version             :    0.1 
// Author              :    Xilin Xia (PhD candidate in Newcastle University)
// Create Time         :    2014/10/04
// Update Time         :    2015/10/25
// ======================================================================================
// Copyright @ Xilin Xia 2015 . All rights reserved.
// ======================================================================================


/*!
\file cuda_SWEs_solver.cu
\brief Source file for component test

*/


#include <iostream>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

//These header files are the primitive types
#include "Flag.h"
#include "Scalar.h"
#include "Vector.h"
#include "cuda_arrays.h"
//These header files are for the fields
#include "mapped_field.h"
#include "cuda_mapped_field.h"
//These header files are for finite volume mesh
#include "mesh_fv_reduced.h"
#include "mesh_interface.h"
#include "cuda_mesh_fv.h"
//These header files are for input and output
#include "gisAsciiMesh_reader.h"
#include "gmsh_reader.h"
#include "field_reader.h"
#include "cuda_simple_writer.h"
#include "cuda_gisascii_writer.h"

//These header files are for shallow water equations advection
#include "cuda_advection_NSWEs.h"
//The header file for gradient
#include "cuda_gradient.h"
//The header file for hessian
#include "cuda_hessian.h"
//The header file for friction
#include "cuda_friction.h"
//The header file for limiter
#include "cuda_limiter.h"
//The header file for field algebra
#include "cuda_field_algebra.h"
//The header file for integrator
#include "cuda_integrators.h"
//The header file for time controllinh
#include "cuda_adaptive_time_control.h"
//The header file for device query
#include "cuda_device_query.h"

//using the name space for GeoClasses
using namespace GC;

int main(){

	deviceQuery();

  //creating an structured finite volume mesh, it is initialized from DEM
  std::shared_ptr<unstructuredFvMesh>  mesh = std::make_shared<unstructuredReducedFvMesh>(gisAsciiMeshReader("input/mesh/DEM.txt"));

  std::cout<<"Read in mesh successfully"<<std::endl;

  //creating mesh on device
  std::shared_ptr<cuUnstructuredFvMesh>  mesh_ptr_dev = std::make_shared<cuUnstructuredFvMesh>(fvMeshQueries(mesh));
  //Read in field data
  fvScalarFieldOnCell z_host(fvMeshQueries(mesh), completeFieldReader("input/field/", "z"));
  fvScalarFieldOnCell h_host(fvMeshQueries(mesh), completeFieldReader("input/field/", "h"));
  fvVectorFieldOnCell hU_host(fvMeshQueries(mesh), completeFieldReader("input/field/", "hU"));
  fvScalarFieldOnCell miu_host(fvMeshQueries(mesh), completeFieldReader("input/field/", "manning"));

  std::cout << "Read in field successfully" << std::endl;

  //h, z, hU
  cuFvMappedField<Scalar, on_cell> z(z_host,mesh_ptr_dev);
  cuFvMappedField<Scalar, on_cell> h(h_host,mesh_ptr_dev);
  cuFvMappedField<Vector, on_cell> hU(hU_host, mesh_ptr_dev);
  cuFvMappedField<Scalar, on_cell> miu(miu_host, mesh_ptr_dev);


  //Velocity
  cuFvMappedField<Vector, on_cell> u(hU, partial);

  //advections
  cuFvMappedField<Scalar, on_cell> h_advection(h, partial);
  cuFvMappedField<Vector, on_cell> hU_advection(hU, partial);

  //friction force
  cuFvMappedField<Vector, on_cell> friction_force(hU, partial);

  //flux
  cuFvMappedField<Scalar, on_halffacet> h_flux(mesh_ptr_dev);
  cuFvMappedField<Vector, on_halffacet> hU_flux(mesh_ptr_dev);

  //old values
  cuFvMappedField<Scalar, on_cell> h_old(h, partial);
  cuFvMappedField<Vector, on_cell> hU_old(hU, partial);
  cuFvMappedField<Scalar, on_cell> h_advection_old(h_advection, partial);
  cuFvMappedField<Vector, on_cell> hU_advection_old(hU_advection, partial);

  //gradients
  cuFvMappedField<Vector, on_cell> h_grad(mesh_ptr_dev);
  cuFvMappedField<Vector, on_cell> z_grad(mesh_ptr_dev);
  cuFvMappedField<Vector, on_cell> eta_grad(mesh_ptr_dev);
  cuFvMappedField<Tensor, on_cell> u_grad(mesh_ptr_dev);

  //gravity
  cuFvMappedField<Scalar, on_cell> gravity(h, partial);
  //setting gravity to single value 9.81
  fv::cuUnaryOn(gravity, [] __device__ (Scalar& a) -> Scalar{return 9.81;}); 

  //hessian
  cuFvMappedField<Tensor, on_cell> z_hessian(mesh_ptr_dev);
  fv::cuHessianCartesian2D(z, z_hessian);

  //centrifugal
  cuFvMappedField<Scalar, on_cell> centrifugal(h, partial);

  //gross vertical acceleration
  cuFvMappedField<Scalar, on_cell> gross_acc(h, partial);

  //modifying factor for gravity
  cuFvMappedField<Scalar, on_cell> phi_b(mesh_ptr_dev);

  //creating uniform friction coefficients field
//  cuFvMappedField<Scalar, on_cell> miu(h, partial);
 // Scalar _miu;
//  std::cout << "Please input friction coefficient" << std::endl;
//  std::cin >> _miu;
//  fv::cuUnaryOn(miu, [=] __device__(Scalar& a) -> Scalar{ return _miu; });

  //Scalar miu1, miu2, W;
  //std::cout << "Please input friction coefficients and wall width" << std::endl;
  //std::cin >> miu1 >> miu2 >> W;


  //surface elevation eta
  cuFvMappedField<Scalar, on_cell> eta(h, partial);

  //calculate the surface elevation
  fv::cuBinary(h, z, eta, [] __device__(Scalar& a, Scalar& b) -> Scalar{ return a + b; });

  Scalar dt_out = 0.5;
  Scalar t_out = 0.0;
  Scalar t_all = 0.0;

  std::cout<<"Please input total time and output time interval"<<std::endl;
  std::cin>>t_all>>dt_out;

  cuAdaptiveTimeControl2D time_controller(0.005, t_all, 0.5);

  int file_id = 0; 
  //write the initial profile
  cuSimpleWriterLowPrecision(z, "z", time_controller.current());
  cuSimpleWriterLowPrecision(h, "h", time_controller.current());
  cuSimpleWriterLowPrecision(eta, "eta", time_controller.current());
  cuSimpleWriterLowPrecision(hU, "hU", time_controller.current());

	//ascii raster writer
	cuGisAsciiWriter raster_writer("input/mesh/DEM.txt");

	//write initial depth
	raster_writer.write(h, "h", time_controller.current());

  file_id++;


  //print current time
  std::cout << time_controller.current() << std::endl;

  double total_runtime = 0.0;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  //Main loop
  do{

     hipEventRecord(start);

    //First step of Runger-Kutta--------------------------------------
    

    //calculate the surface elevation
    fv::cuBinary(h, z, eta, [] __device__ (Scalar& a, Scalar& b) -> Scalar{return a + b;});

    //calculate the velocity
    auto divide = [] __device__ (Vector& a, Scalar& b) ->Vector{
      if(b >= 1e-10){
        return a/b;
      }else{
        return Vector(0.0);
      }
    };

    fv::cuBinary(hU, h, u, divide); 

    //store the old values, may need to be replaced by assining operators
    fv::cuBinaryOn(h_old, h, [] __device__ (Scalar& a, Scalar& b) -> Scalar{return b;});
    fv::cuBinaryOn(hU_old, hU, [] __device__ (Vector& a, Vector& b) -> Vector{return b;});

    //calculate gradient
    fv::cuGradient(h, h_grad);
    fv::cuGradient(eta, eta_grad);
    fv::cuGradient(u, u_grad);

    //Limiting gradient
    fv::cuGradientLimiterCartesian(h, h_grad);
    fv::cuGradientLimiterCartesian(eta, eta_grad);
    fv::cuGradientLimiterCartesian(u, u_grad);

    //modifying the gravity
    fv::cuGradient(z, z_grad);
    fv::cuGradientLimiterCartesian(z, z_grad);
    //setting gravity to single value 9.81
    fv::cuUnaryOn(gravity, [] __device__(Scalar& a) -> Scalar{ return 9.81; });
	//calculating modifying factor
	  //fv::cuBinary(z_grad, eta_grad, phi_b, [] __device__(Vector& a, Vector& b) -> Scalar{ return fmax(0.0, fmin(1.0, 1 / (1.0 + dot(a, b)))); });
	  //fv::cuBinary(phi_b, gravity, gravity, [] __device__(Scalar& a, Scalar& b) -> Scalar{ return a*b; });
    fv::cuBinary(z_grad, gravity, gravity, [] __device__(Vector& a, Scalar& b) -> Scalar{ return b / (1.0 + dot(a, a)); });

    //calculate centrifugal force
    fv::cuBinary(u, z_hessian, centrifugal, [] __device__(Vector& a, Tensor& b) -> Scalar{ return dot(dot(b, a), a); });
    fv::cuBinary(z_grad, centrifugal, centrifugal, [] __device__(Vector& a, Scalar& b) -> Scalar{ return b / (1.0 + dot(a, a)); });
    fv::cuBinary(gravity, centrifugal, centrifugal, [] __device__(Scalar& a, Scalar& b) -> Scalar{ return fmin(20.0*a, b); });

    //calculate advection
 //   fv::cuAdvectionNSWEs2ndRobust(gravity, h, z, u, h_grad, eta_grad, u_grad, h_advection, hU_advection);
    fv::cuAdvectionNSWEs2ndRobustCurv(gravity, centrifugal, h, z, u, h_grad, eta_grad, u_grad, h_advection, hU_advection);


    //multiply advection with -1
    fv::cuUnaryOn(h_advection, [] __device__ (Scalar& a) -> Scalar{return -1.0*a;});
    fv::cuUnaryOn(hU_advection, [] __device__ (Vector& a) -> Vector{return -1.0*a;});


    //integration
    fv::cuEulerIntegrator(h, h_advection, time_controller.dt(), time_controller.current());
    fv::cuEulerIntegrator(hU, hU_advection, time_controller.dt(), time_controller.current());

    //store the old values, may need to be replaced by assining operators
    fv::cuBinaryOn(h_advection_old, h_advection, [] __device__ (Scalar& a, Scalar& b) -> Scalar{return b;});
    fv::cuBinaryOn(hU_advection_old, hU_advection, [] __device__ (Vector& a, Vector& b) -> Vector{return b;});

    auto filter = [] __device__ (Vector& a, Scalar& b) ->Vector{
      if(b <= 1e-6){
        return Vector(0.0);
      }else{
        return a;
      }
    };

    fv::cuBinaryOn(hU, h, filter);

    auto filter1 = [] __device__ (Scalar& a) ->Scalar{
      if(a <= 1e-6){
        return 0.0;
      }else{
        return a;
      }
    };

    fv::cuUnaryOn(h, filter1);

    //Second step of Runger-Kutta--------------------------------------


    //calculate the surface elevation
    fv::cuBinary(h, z, eta, [] __device__ (Scalar& a, Scalar& b) -> Scalar{return a + b;});

    //calculate the velocity
    fv::cuBinary(hU, h, u, divide); 

    //calculate gradient
    fv::cuGradient(h, h_grad);
    fv::cuGradient(eta, eta_grad);
    fv::cuGradient(u, u_grad);

    //Limiting gradient
    fv::cuGradientLimiterCartesian(h, h_grad);
    fv::cuGradientLimiterCartesian(eta, eta_grad);
    fv::cuGradientLimiterCartesian(u, u_grad);

    //modifying the gravity
    fv::cuGradient(z, z_grad);
    fv::cuGradientLimiterCartesian(z, z_grad);
    //setting gravity to single value 9.81
    fv::cuUnaryOn(gravity, [] __device__(Scalar& a) -> Scalar{ return 9.81; });
	//calculating modifying factor
	  //fv::cuBinary(z_grad, eta_grad, phi_b, [] __device__(Vector& a, Vector& b) -> Scalar{ return fmax(0.0, fmin(1.0, 1 / (1.0 + dot(a, b)))); });
	  //fv::cuBinary(phi_b, gravity, gravity, [] __device__(Scalar& a, Scalar& b) -> Scalar{ return a*b; });
    fv::cuBinary(z_grad, gravity, gravity, [] __device__(Vector& a, Scalar& b) -> Scalar{ return b / (1.0 + dot(a, a)); });

    //calculate centrifugal force
    fv::cuBinary(u, z_hessian, centrifugal, [] __device__(Vector& a, Tensor& b) -> Scalar{ return dot(dot(b, a), a); });
    fv::cuBinary(z_grad, centrifugal, centrifugal, [] __device__(Vector& a, Scalar& b) -> Scalar{ return b / (1.0 + dot(a, a)); });
    fv::cuBinary(gravity, centrifugal, centrifugal, [] __device__(Scalar& a, Scalar& b) -> Scalar{ return fmin(20.0*a, b); });

    //calculate advection
  //  fv::cuAdvectionNSWEs2ndRobust(gravity, h, z, u, h_grad, eta_grad, u_grad, h_advection, hU_advection);
    fv::cuAdvectionNSWEs2ndRobustCurv(gravity, centrifugal, h, z, u, h_grad, eta_grad, u_grad, h_advection, hU_advection);


    //multiply advection with -1
    fv::cuUnaryOn(h_advection, [] __device__ (Scalar& a) -> Scalar{return -1.0*a;});
    fv::cuUnaryOn(hU_advection, [] __device__ (Vector& a) -> Vector{return -1.0*a;});


    //store old values for friction calculation
    fv::cuBinaryOn(h, h_old, [] __device__(Scalar& a, Scalar& b) -> Scalar{ return b; });

    //integration
    fv::cuEulerIntegrator(h_old, h_advection_old, time_controller.dt() / 2.0, time_controller.current());
    fv::cuEulerIntegrator(hU_old, hU_advection_old, time_controller.dt() / 2.0, time_controller.current());
    fv::cuEulerIntegrator(h_old, h_advection, time_controller.dt() / 2.0, time_controller.current());
    fv::cuEulerIntegrator(hU_old, hU_advection, time_controller.dt() / 2.0, time_controller.current());

    //Write updated values hU
    fv::cuBinaryOn(hU, hU_old, [] __device__ (Vector& a, Vector& b) -> Vector{return b;});


    fv::cuBinaryOn(hU, h, filter);
    fv::cuUnaryOn(h, filter1);

    //add centrifugal force to gravity to obtain gross vertical acceleration
    fv::cuBinary(gravity, centrifugal, gross_acc, [] __device__(Scalar& a, Scalar& b) -> Scalar{ return a + b; });

    //Calculating friction, use old h for depth, new hU for direction
    fv::cuFrictionMCBalanced(time_controller.dt(), gravity, miu, h, hU, z_grad, friction_force);
    //fv::cuFrictionMCWithWall(time_controller.dt(), gross_acc, miu1, miu2, W, h, hU, z_grad, friction_force);
    fv::cuEulerIntegrator(hU, friction_force, time_controller.dt(), time_controller.current());

    //Write updated values h
    fv::cuBinaryOn(h, h_old, [] __device__(Scalar& a, Scalar& b) -> Scalar{ return b; });


    //forwarding the time
    time_controller.forward();
    time_controller.updateByCFL(gravity,h,hU);

    //print current time
    std::cout << time_controller.current() << std::endl;

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float elapsed_time = 0.0;
    hipEventElapsedTime(&elapsed_time, start, stop);
    total_runtime += elapsed_time;

    if (time_controller.current() >= t_out){
      cuSimpleWriterLowPrecision(h, "h", t_out);
      cuSimpleWriterLowPrecision(hU, "hU", t_out);
      cuSimpleWriterLowPrecision(eta, "elev", t_out);

	  raster_writer.write(eta, "elev", t_out);
	  raster_writer.write(h, "h", t_out);
      file_id++;
      t_out += dt_out;
    }


  } while (!time_controller.is_end());

  std::cout << "Total runtime " << total_runtime << "ms" << std::endl;

  return 0;

}

